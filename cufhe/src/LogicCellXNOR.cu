#include "LogicCellXNOR.hpp"

LogicCellXNOR::LogicCellXNOR(
    int id,
    int pri,
    bool isCipher) : Logic(id, pri, isCipher) {
    Type = "XNOR";
}

void LogicCellXNOR::Prepare() {
    if (input.size() != 2) {
        throw std::runtime_error("Input is not assigned");
    }
    if (output.size() == 0) {
        throw std::runtime_error("Output is not assigned");
    }

    InputCount = input.size();
    ReadyInputCount = 0;
}

void LogicCellXNOR::Execute(cufhe::Stream stream, bool reset) {
    cufhe::gXnor(*value, *input.at(0)->value, *input.at(1)->value, stream);
    executed = true;
}

void LogicCellXNOR::Execute(bool reset) {
    res = (~(input.at(0)->res ^ input.at(1)->res)) & 0x1;
    executed = true;
}

bool LogicCellXNOR::NoticeInputReady() {
    ReadyInputCount++;
    if (ReadyInputCount > InputCount) {
        throw std::runtime_error("[XNOR] ReadyInputCount is invalid");
    }
    return InputCount == ReadyInputCount;
}

void LogicCellXNOR::AddInput(Logic *logic) {
    if (input.size() > 1) {
        throw std::runtime_error("Input is already assigned");
    }
    input.push_back(logic);
}

void LogicCellXNOR::AddOutput(Logic *logic) {
    output.push_back(logic);
}

bool LogicCellXNOR::Tick() {
    executable = false;
    executed = false;
    ReadyInputCount = 0;
    return executable;
}
