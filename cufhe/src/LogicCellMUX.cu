#include "LogicCellMUX.hpp"

LogicCellMUX::LogicCellMUX(
    int id,
    int pri,
    bool isCipher) : Logic(id, pri, isCipher) {
    Type = "MUX";
}

void LogicCellMUX::Prepare() {
    if (input.size() != 3) {
        throw std::runtime_error("Input is not assigned");
    }
    if (output.size() == 0) {
        throw std::runtime_error("Output is not assigned");
    }

    InputCount = input.size();
    ReadyInputCount = 0;
}

void LogicCellMUX::Execute(cufhe::Stream stream, bool reset) {
    cufhe::gMux(*value, *input.at(2)->value, *input.at(1)->value, *input.at(0)->value, stream);
    executed = true;
}

void LogicCellMUX::Execute(bool reset) {
    if (input.at(2)->res == 0) {
        res = input.at(0)->res;
    } else if (input.at(2)->res == 1) {
        res = input.at(1)->res;
    } else {
        throw std::runtime_error("invalid select signal");
    }
    executed = true;
}

bool LogicCellMUX::NoticeInputReady() {
    ReadyInputCount++;
    if (ReadyInputCount > InputCount) {
        throw std::runtime_error("[MUX] ReadyInputCount is invalid");
    }
    return InputCount == ReadyInputCount;
}

void LogicCellMUX::AddInput(Logic *logic) {
    if (input.size() > 2) {
        throw std::runtime_error("Input is already assigned");
    }
    input.push_back(logic);
}

void LogicCellMUX::AddOutput(Logic *logic) {
    output.push_back(logic);
}

bool LogicCellMUX::Tick() {
    executable = false;
    executed = false;
    ReadyInputCount = 0;
    return executable;
}
